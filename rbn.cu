#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define cinf(n,x) for(int i = 0;i<n;i++)cin >>x[i];
using namespace std;

__global__ void add_joutyou(int* d_a, int* d_b, int* d_c, int* d_s, int n){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n) {
		/*一番下の桁の被加数と加数から中間和と中間桁上げの場合分けを以下のように行った。
	二番目以降の桁は一つ下の桁の被加数と加数の値によって分岐するが、この桁だけは分岐しないため、他の桁とは別で処理を行った。*/
		if (i == 0) {
			if (d_a[0] == 1 && d_b[0] == 1) {
				d_c[0] = 1;
				d_s[0] = 0;
			}
			else if ((d_a[0] == 1 && d_b[0] == 0) || (d_a[0] == 0 && d_b[0] == 1)) {
				d_c[0] = 0;
				d_s[0] = 1;
			}
			else if (d_a[0] == 0 && d_b[0] == 0) {
				d_c[0] = 0;
				d_s[0] = 0;
			}
			else if ((d_a[0] == -1 && d_b[0] == 1) || (d_a[0] == 1 && d_b[0] == -1)) {
				d_c[0] = 0;
				d_s[0] = 0;
			}
			else if ((d_a[0] == -1 && d_b[0] == 0) || (d_a[0] == 0 && d_b[0] == -1)) {
				d_c[0] = 0;
				d_s[0] = -1;
			}
			else if (d_a[0] == -1 && d_b[0] == -1) {
				d_c[0] = -1;
				d_s[0] = 0;
			}
		}
		/*2桁目以降の桁の被加数と加数から中間和と中間桁上げの場合分けを以下のように行った。
	一桁目とは異なり、前の桁の被加数と加数の値によって分岐することがある。*/
		else {
			for (int i = 1; i < n; i++) {
				if (d_a[i] == 1 && d_b[i] == 1) {
					d_c[i] = 1;
					d_s[i] = 0;
				}
				else if ((d_a[i] == 1 && d_b[i] == 0) || (d_a[i] == 0 && d_b[i] == 1)) {
					if (d_a[i - 1] >= 0 && d_b[i - 1] >= 0) {
						d_c[i] = 1;
						d_s[i] = -1;
					}
					else {
						d_c[i] = 0;
						d_s[i] = 1;
					}
				}
				else if (d_a[i] == 0 && d_b[i] == 0) {
					d_c[i] = 0;
					d_s[i] = 0;
				}
				else if ((d_a[i] == 1 && d_b[i] == -1) || (d_a[i] == -1 && d_b[i] == 1)) {
					d_c[i] = 0;
					d_s[i] = 0;
				}
				else if ((d_a[i] == 0 && d_b[i] == -1) || (d_a[i] == -1 && d_b[i] == 0)) {
					if (d_a[i - 1] >= 0 && d_b[i - 1] >= 0) {
						d_c[i] = 0;
						d_s[i] = -1;
					}
					else {
						d_c[i] = -1;
						d_s[i] = 1;
					}
				}
				else if (d_a[i] == -1 && d_b[i] == -1) {
					d_c[i] = -1;
					d_s[i] = 0;
				}
			}
		}
	}

}
int main(int argc,char*argv[]) {
	//桁数nを入力
	int n;
	cin >> n;

	//CPUの動的メモリの確保
	int* a, * b, * c, * s;
	a = (int*)malloc(n * sizeof(int));
	b = (int*)malloc(n * sizeof(int));
	c = (int*)malloc(n * sizeof(int));
	s = (int*)malloc(n * sizeof(int));

	//GPUの動的メモリの確保
	int* d_a,* d_b,* d_c,* d_s;
	hipMalloc(&d_a, n * sizeof(int));
	hipMalloc(&d_b, n * sizeof(int));
	hipMalloc(&d_c, n * sizeof(int));
	hipMalloc(&d_s, n * sizeof(int));

	
	//n桁の被加数a[n]を入力。但し-1から1まで
	cinf(n, a);

	//n桁の加数b[n]を入力。但し-1から1まで
	cinf(n, b);

	//CPUからGPUにメモリを移動させる
	hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

	//被加数と加数から中間和と中間桁上げを出力する関数を生成
	add_joutyou << < (n + 256 - 1) / 256, 256 >> > (d_a, d_b, d_c, d_s, n);

	//GPUからCPUにメモリを戻す
	hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(s, d_s, n * sizeof(int), hipMemcpyDeviceToHost);

	//全ての桁の中間桁上げを出力
	cout << "中間桁上げ" << endl;
	for (int i = 0; i < n; i++) {
		cout << c[i] << " ";
	}
	cout << endl;
	//全ての桁の中間和を出力
	cout << "中間和" << endl;
	for (int i = 0; i < n; i++) {
		cout << s[i] << " ";
	}
	cout << endl;
	//全ての桁の和を出力
	cout << "和" << endl;
	cout << s[0] << " ";
	for (int i = 1; i < n; i++) {
		cout << s[i] + c[i - 1] << " ";
	}
	cout << c[n - 1] << endl;

	//CPu上のメモリの開放
	free(a);
	free(b);
	free(c);
	free(s);
	//GPU上のメモリの開放
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_s);
}