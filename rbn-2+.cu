#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define cinf(n,x) for(int i = 0;i<n;i++)cin >>x[i];
using namespace std;

__global__ void add_joutyou(int* d_a, int* d_b, int* d_c, int* d_s, int n) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < n) {
		/*�S�Ă̔�����Ɖ������璆�Ԙa�ƒ��Ԍ��グ�̏ꍇ�������ȉ��̂悤�ɍs�����B*/
		for (int i = 0; i < n; i++) {
			if (d_a[i] == 1 && d_b[i] == 1) {
				d_c[i] = 1;
				d_s[i] = 0;
			}
			else if ((d_a[i] == 1 && d_b[i] == 0) || (d_a[i] == 0 && d_b[i] == 1)) {
				d_c[i] = 1;
				d_s[i] = -1;
			}
			else if (d_a[i] == 0 && d_b[i] == 0) {
				d_c[i] = 0;
				d_s[i] = 0;
			}
			else if (d_a[i] == -1 && d_b[i] == 1) {
				d_c[i] = 0;
				d_s[i] = 0;
			}
			else if (d_a[i] == -1 && d_b[i] == 0) {
				d_c[i] = 0;
				d_s[i] = -1;
			}
			else if (d_a[i] == -2 && d_b[i] == 1) {
				d_c[i] = 0;
				d_s[i] = -1;
			}
			else if (d_a[i] == -2 && d_b[i] == 0) {
				d_c[i] = -1;
				d_s[i] = 0;
			}
		}
	}
}

int main(int argc, char* argv[]) {
	//����n�����
	int n;
	cin >> n;

	//CPU�̓��I�������̊m��
	int* a, * b, * c, * s;
	a = (int*)malloc(n * sizeof(int));
	b = (int*)malloc(n * sizeof(int));
	c = (int*)malloc(n * sizeof(int));
	s = (int*)malloc(n * sizeof(int));

	//GPU�̓��I�������̊m��
	int* d_a, * d_b, * d_c, * d_s;
	hipMalloc(&d_a, n * sizeof(int));
	hipMalloc(&d_b, n * sizeof(int));
	hipMalloc(&d_c, n * sizeof(int));
	hipMalloc(&d_s, n * sizeof(int));


	//n���̔����a[n]����́B�A��-2����1�܂�
	cinf(n, a);

	//n���̉���b[n]����́B�A��0����1�܂�
	cinf(n, b);

	//CPU����GPU�Ƀ��������ړ�������
	hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

	//������Ɖ������璆�Ԙa�ƒ��Ԍ��グ���o�͂���֐��𐶐�
	add_joutyou << < (n + 256 - 1) / 256, 256 >> > (d_a, d_b, d_c, d_s, n);

	//GPU����CPU�Ƀ�������߂�
	hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(s, d_s, n * sizeof(int), hipMemcpyDeviceToHost);

	//�S�Ă̌��̒��Ԍ��グ���o��
	cout << "���Ԍ��グ" << endl;
	for (int i = 0; i < n; i++) {
		cout << c[i] << " ";
	}
	cout << endl;
	//�S�Ă̌��̒��Ԙa���o��
	cout << "���Ԙa" << endl;
	for (int i = 0; i < n; i++) {
		cout << s[i] << " ";
	}
	cout << endl;
	//�S�Ă̌��̘a���o��
	cout << "�a" << endl;
	cout << s[0] << " ";
	for (int i = 1; i < n; i++) {
		cout << s[i] + c[i - 1] << " ";
	}
	cout << c[n - 1] << endl;

	//CPu��̃������̊J��
	free(a);
	free(b);
	free(c);
	free(s);
	//GPU��̃������̊J��
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipFree(d_s);
}